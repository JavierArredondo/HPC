#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>

// CUDA kernel. Cada thread ejecuta la operación sobrte un elemencto de c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Obtención del Id global
    int id = blockIdx.x*blockDim.x+threadIdx.x;
     // Nos aseguramos de no salir de los bordes
    if (id < n)
        c[id] = a[id] + b[id];
    printf("I am thread (%d, %d) in block (%d, %d). Global thread ID = %d\n",threadIdx.y, threadIdx.x, blockIdx.y, blockIdx.x);
    
}
 
__host__ int main( int argc, char* argv[] )
{
     
    // Tamaño de los vectores
    int n = 100000;
 
    // Vectores de entrada al host (CPU)
    double *h_a;
    double *h_b;
    // Vector de salida del host
    double *h_c;
 
    // Vector de entrada del device (GPU)
    double *d_a;
    double *d_b;
    // Vector de salida del device
    double *d_c;
 
    // Size, in bytes, of each vector
    //size_t bytes = n*sizeof(double);
 
    // Se asigna memoria para cada vector del host
    h_a = (double*)malloc(n*sizeof(double));
    h_b = (double*)malloc(n*sizeof(double));
    h_c = (double*)malloc(n*sizeof(double));
 
    // Se asigna memoria para cada vector del device
    hipMalloc(&d_a, n*sizeof(double));
    hipMalloc(&d_b, n*sizeof(double));
    hipMalloc(&d_c, n*sizeof(double));
 
    int i;
    // Se inicializa los vectores del host
    for( i = 0; i < n; i++ ) {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
    }
 
    // Se copia el vector del host al vector del device
    hipMemcpy( d_a, h_a, n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, n*sizeof(double), hipMemcpyHostToDevice);
 
    int blockSize, gridSize;
 
    // Número de threads en cada bloque
    blockSize = 1024;
 
    // Número de bloques en la grilla
    gridSize = (int)ceil((float)n/blockSize);
    printf("%i\n", gridSize);
 
    // Se ejecuta el kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
 
    // Se copia el vector resultante del device al host
    hipMemcpy( h_c, d_c, n*sizeof(double), hipMemcpyDeviceToHost );

    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    printf("final : %f\n", sum/n);
 
    // Se libera la memoria del device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    // Se libera la memoria del host
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}